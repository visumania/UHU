//%%writefile parallel.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef int DATA_TYPE;
#define MAX_INT_VAL 10
#define OK 1

void printMatrix(DATA_TYPE * matrix, int rows, int cols); /* function prototype */
void loadRandMatrix(DATA_TYPE ** matrix, int rows, int cols);

__global__ void multiAB(DATA_TYPE* matrix_A, DATA_TYPE* matrix_B, DATA_TYPE* matrix_C, int nrowA, int ncolA, int ncolB){
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if(row<nrowA && col<ncolB){
    DATA_TYPE add = 0;

    for(int k=0 ; k<ncolA ; k++){
      add += matrix_A[row*ncolA+k] * matrix_B[col+k*ncolB];
    }
      
  }
}

int main(void){
	srand((unsigned) time(NULL)); /* random seed */
	double start_time, stop_time; /* for Windows, Linux */
	
	
	DATA_TYPE * A_matrix = NULL; //Pointer to matrix in the host
	DATA_TYPE * B_matrix = NULL; //Pointer to matrix in the host
	DATA_TYPE * C_matrix = NULL; //Pointer to matrix in the host
 
  DATA_TYPE * A_matrix_d = NULL; //Pointer to matrix in the device
	DATA_TYPE * B_matrix_d = NULL; //Pointer to matrix in the device
	DATA_TYPE * C_matrix_d = NULL; //Pointer to matrix in the device
 
	int n, m, p;
	
	/* setting matrices params - n, m and p */
	printf("Enter the number of A rows = "); scanf("%d",&m);
    printf("Enter the number of A cols ( = B rows) = "); scanf("%d",&n);
    printf("Enter the number of B cols = "); scanf("%d",&p);

  int N = m*p;
  size_t size = N*sizeof(DATA_TYPE);
	
	/* create all matrices for C = AxB */
	
	A_matrix = (DATA_TYPE *)malloc(size);
	B_matrix = (DATA_TYPE *)malloc(size);
	C_matrix = (DATA_TYPE *)malloc(size);	
				
	//Load matrix A
  for(int i=0 ; i<m ; i++){
      for(int j=0 ; j<n ; j++){
          A_matrix[i*n+j] = rand() % MAX_INT_VAL;
      }
  }

  //Load matrix B
	for(int i=0 ; i<n ; i++){
      for(int j=0 ; j<p ; j++){
          B_matrix[i*p+j] = rand() % MAX_INT_VAL;
      }
  }

	//For debuging
	//printMatrix(A_matrix, m, n);
	//printMatrix(B_matrix, n, p);
 
  //Alloc space for device copies
  hipMalloc((void **) &A_matrix_d, size);
  hipMalloc((void **) &B_matrix_d, size);
  hipMalloc((void **) &C_matrix_d, size);
 
  //Copy to device
  hipMemcpy(A_matrix_d, A_matrix, size, hipMemcpyHostToDevice);
  hipMemcpy(B_matrix_d, B_matrix, size, hipMemcpyHostToDevice);

  //Threads per block
  int BLOCK_SIZE = 16;

  //Blocks in each dimension
  int GRID_SIZE = (int)ceil(N/BLOCK_SIZE);

  dim3 grid(GRID_SIZE,GRID_SIZE);
  dim3 threads(BLOCK_SIZE,BLOCK_SIZE);

	start_time = clock();  /* time measurement */
		
	multiAB<<<grid,threads>>>(A_matrix_d,B_matrix_d,C_matrix_d, m,n,p);  
  hipDeviceSynchronize();
	
	stop_time = clock(); /* time measurement */
	
	hipMemcpy(C_matrix, C_matrix_d, size, hipMemcpyDeviceToHost);
	
	//printMatrix(C_matrix, m, p); 
					
	free(A_matrix);
  free(B_matrix);
  free(C_matrix);

  hipFree(A_matrix_d);
  hipFree(B_matrix_d);
  hipFree(C_matrix_d);

	printf("\nElapsed time of matrices multiplication: %f [seconds]\n",
		  (stop_time-start_time)/CLOCKS_PER_SEC);
	
	puts("\nFinished");

	return 0;
}

void printMatrix(DATA_TYPE * matrix, int rows, int cols)
{
	int i, j;
	
	printf("\nPrint matrix:");
	
	for(i=0; i<rows; i++)
	{
		printf("\n");
		for(j=0; j<cols; j++)
		    printf("%3d ", matrix[i*cols+j]); //the same as matrix[i][j] 
	}
	printf("\n");
	return;
}

void loadRandMatrix(DATA_TYPE ** matrix, int rows, int cols)
{
	int i, j;
	
	printf("\nLoad matrix:");
	
	for(i=0; i<rows; i++)
	{
		for(j=0; j<cols; j++)
			matrix[i][j] = rand() % MAX_INT_VAL;
	}
	return;
}